#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "parallelHistogram.h"
#include "parallelHistogramDriver.h"
#include "types/constants.h"

extern "C" float parallelHistogramDriver(int* data_h, int length, int* hist_h,
                                         enum parallelHistogramKernelToUse kernel_to_use, int iters)
{
    int *data_d, *hist_d;
    dim3 dimBlock, dimGrid;

    int hist_d_size = NUM_BINS * sizeof(int);
    if (kernel_to_use == kPrivatized)
    {
        hist_d_size *= ceil((float)length / NUM_BINS);
    }
    hipMalloc((void**)&data_d, length * sizeof(int));
    hipMalloc((void**)&hist_d, hist_d_size);

    hipMemcpy(data_d, data_h, length * sizeof(int), hipMemcpyHostToDevice);

    float time;
    float total_time = 0.0f;
    hipEvent_t start, stop;

    for (int iter = 0; iter < iters; ++iter)
    {
        hipMemset(hist_d, 0, hist_d_size);
        switch (kernel_to_use)
        {
            case kBasic:
                dimBlock = dim3(TILE_WIDTH, 1, 1);
                dimGrid = dim3(ceil((float)length / dimBlock.x), 1, 1);
                hipEventCreate(&start);
                hipEventCreate(&stop);
                hipEventRecord(start, 0);
                basicParallelHistogram<<<dimGrid, dimBlock>>>(data_d, length, hist_d);
                break;
            case kPrivatized:
                dimBlock = dim3(TILE_WIDTH, 1, 1);
                dimGrid = dim3(ceil((float)length / dimBlock.x), 1, 1);
                hipEventCreate(&start);
                hipEventCreate(&stop);
                hipEventRecord(start, 0);
                privatizedParallelHistogram<<<dimGrid, dimBlock>>>(data_d, length, hist_d);
                break;
            case kPrivatizedWithSharedMemory:
                dimBlock = dim3(TILE_WIDTH, 1, 1);
                dimGrid = dim3(ceil((float)length / dimBlock.x), 1, 1);
                hipEventCreate(&start);
                hipEventCreate(&stop);
                hipEventRecord(start, 0);
                privatizedWithSharedMemoryParallelHistogram<<<dimGrid, dimBlock>>>(data_d, length,
                                                                                   hist_d);
                break;
            case kCoarsening:
                dimBlock = dim3(TILE_WIDTH, 1, 1);
                dimGrid = dim3(ceil(((float)length / CFACTOR) / dimBlock.x), 1, 1);
                hipEventCreate(&start);
                hipEventCreate(&stop);
                hipEventRecord(start, 0);
                coarseningParallelHistogram<<<dimGrid, dimBlock>>>(data_d, length, hist_d);
                break;
            case kCoarseningWithCoalescedAccess:
                dimBlock = dim3(TILE_WIDTH, 1, 1);
                dimGrid = dim3(ceil(((float)length / CFACTOR) / dimBlock.x), 1, 1);
                hipEventCreate(&start);
                hipEventCreate(&stop);
                hipEventRecord(start, 0);
                coarseningWithCoalescedAccessParallelHistogram<<<dimGrid, dimBlock>>>(
                    data_d, length, hist_d);
                break;
            case kAggregated:
                dimBlock = dim3(TILE_WIDTH, 1, 1);
                dimGrid = dim3(ceil(((float)length / CFACTOR) / dimBlock.x), 1, 1);
                hipEventCreate(&start);
                hipEventCreate(&stop);
                hipEventRecord(start, 0);
                aggregatedParallelHistogram<<<dimGrid, dimBlock>>>(data_d, length, hist_d);
                break;
            case kNumKernels:
            default:
                break;
        }
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        total_time += time;
    }

    hipError_t err = hipMemcpy(hist_h, hist_d, NUM_BINS * sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d.\n", hipGetErrorString(err), __FILE__, __LINE__);
    }
    hipFree(hist_d);
    hipFree(data_d);

    return total_time;
}
