#include "hip/hip_runtime.h"
/// @file bezier.cu
/// @brief CUDA implementation of Bezier curve tessellation.

#include <hip/hip_runtime.h>
#include "bezier.h"
#include "bezier_utils.h"
#include "types/constants.h"
#include "types/types.h"

__global__ void ComputeBezierLinesBasic(BezierLineFixedSize* lines, const int num_lines)
{
    const unsigned int b_idx{blockIdx.x};
    if (b_idx < num_lines)
    {
        auto& line = lines[b_idx];
        const auto n_tess_points = ComputeNumberOfTessPoints(lines[b_idx]);
        line.num_vertices = n_tess_points;

        for (int inc{0}; inc < n_tess_points; inc += blockDim.x)
        {
            int idx = inc + threadIdx.x;
            if (idx < n_tess_points)
            {
                const float t = static_cast<float>(idx) / (n_tess_points - 1);
                const float t2 = t * t;
                const float one_minus_t = 1.F - t;
                const float one_minus_t2 = one_minus_t * one_minus_t;
                line.vertex_pos[idx] = one_minus_t2 * line.CP[0] +
                                       2.F * one_minus_t * t * line.CP[1] + t2 * line.CP[2];
            }
        }
    }
}

__global__ void ComputeBezierLinesDynamic(BezierLineDynamic* lines, const int num_lines)
{
    int l_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (l_idx < num_lines)
    {
        const auto n_tess_points = ComputeNumberOfTessPoints(lines[l_idx]);
        lines[l_idx].num_vertices = n_tess_points;
        if (!hipMalloc((void**)&lines[l_idx].vertex_pos, n_tess_points * sizeof(float2)))
        {
            ComputeBezierLineDynamic<<<ceil(static_cast<float>(n_tess_points) / 32.F), 32>>>(
                l_idx, lines, n_tess_points);
        }
    }
}

__global__ void ComputeBezierLineDynamic(const int line_idx, BezierLineDynamic* lines,
                                         const int n_points)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n_points)
    {
        const float t = static_cast<float>(idx) / (n_points - 1);
        const float t2 = t * t;
        const float one_minus_t = 1.F - t;
        const float one_minus_t2 = one_minus_t * one_minus_t;
        lines[line_idx].vertex_pos[idx] = one_minus_t2 * lines[line_idx].CP[0] +
                                          2.F * one_minus_t * t * lines[line_idx].CP[1] +
                                          t2 * lines[line_idx].CP[2];
    }
}

__global__ void FreeVertexMem(BezierLineDynamic* lines, const int num_lines)
{
    int l_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (l_idx < num_lines)
    {
        hipFree(lines[l_idx].vertex_pos);
    }
}