#include "hip/hip_runtime.h"
/// @file bfs_driver.cu
/// @brief Definition of driver function declared in bfs_driver.h.

#include <stdio.h>
#include <algorithm>
#include "bfs.h"
#include "bfs_driver.h"
#include "types/adjacency_matrix.h"
#include "types/constants.h"
#include "types/graph_coo.h"

float EdgeCentricDriver(AdjacencyMatrix& adj_matrix, int* result_h, const int iters)
{
    GraphCoo graph_coo_h{adj_matrix.ToCoo()};
    GraphCoo graph_coo_h_to_copy_to_d{};
    graph_coo_h_to_copy_to_d.num_edges = graph_coo_h.num_edges;
    hipMalloc((void**)&graph_coo_h_to_copy_to_d.src, graph_coo_h.num_edges * sizeof(int));
    hipMalloc((void**)&graph_coo_h_to_copy_to_d.dst, graph_coo_h.num_edges * sizeof(int));
    hipMalloc((void**)&graph_coo_h_to_copy_to_d.val, graph_coo_h.num_edges * sizeof(int));
    hipMemcpy(graph_coo_h_to_copy_to_d.src, graph_coo_h.src, graph_coo_h.num_edges * sizeof(int),
               hipMemcpyHostToDevice);
    hipMemcpy(graph_coo_h_to_copy_to_d.dst, graph_coo_h.dst, graph_coo_h.num_edges * sizeof(int),
               hipMemcpyHostToDevice);
    hipMemcpy(graph_coo_h_to_copy_to_d.val, graph_coo_h.val, graph_coo_h.num_edges * sizeof(int),
               hipMemcpyHostToDevice);

    GraphCoo* graph_coo_d{};
    hipMalloc((void**)&graph_coo_d, sizeof(GraphCoo));
    hipMemcpy(graph_coo_d, &graph_coo_h_to_copy_to_d, sizeof(GraphCoo), hipMemcpyHostToDevice);
    int* result_d{};

    hipMalloc((void**)&result_d, adj_matrix.GetN() * sizeof(int));

    float time{};
    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    for (int i = 0; i < iters; ++i)
    {
        hipMemset(result_d, -1, adj_matrix.GetN() * sizeof(int));
        hipMemset(result_d, 0, sizeof(int));
        dim3 block_dim{SECTION_SIZE, 1, 1};
        dim3 grid_dim{static_cast<unsigned int>(
                          ceil(static_cast<float>(graph_coo_h.num_edges) / SECTION_SIZE)),
                      1, 1};
        int new_vertex_visited{0};
        int curr_level{1};
        do
        {
            new_vertex_visited = 0;
            hipMemcpy(&new_vertex_visited, &new_vertex_visited, sizeof(int),
                       hipMemcpyHostToDevice);
            EdgeCentricBFS<<<grid_dim, block_dim>>>(graph_coo_d, result_d, &new_vertex_visited,
                                                    curr_level);
            hipMemcpy(&new_vertex_visited, &new_vertex_visited, sizeof(int),
                       hipMemcpyDeviceToHost);
            ++curr_level;
        } while (new_vertex_visited != 0);

        hipMemcpy(result_h, result_d, adj_matrix.GetN() * sizeof(int), hipMemcpyDeviceToHost);
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(graph_coo_d);
    hipFree(graph_coo_h_to_copy_to_d.src);
    hipFree(graph_coo_h_to_copy_to_d.dst);
    hipFree(graph_coo_h_to_copy_to_d.val);
    hipFree(result_d);

    delete[] graph_coo_h.src;
    delete[] graph_coo_h.dst;
    delete[] graph_coo_h.val;

    return time;
}

float VertexCentricPushDriver(AdjacencyMatrix& adj_matrix, int* result_h, const int iters)
{
    GraphCsr graph_csr_h{adj_matrix.ToCsr()};
    GraphCsr graph_csr_h_to_copy_to_d{};
    graph_csr_h_to_copy_to_d.n = graph_csr_h.n;
    const int num_edges{graph_csr_h.row_ptrs[graph_csr_h.n]};
    hipMalloc((void**)&graph_csr_h_to_copy_to_d.row_ptrs, (graph_csr_h.n + 1) * sizeof(int));
    hipMalloc((void**)&graph_csr_h_to_copy_to_d.col_idx, num_edges * sizeof(int));
    hipMalloc((void**)&graph_csr_h_to_copy_to_d.val, num_edges * sizeof(int));
    hipMemcpy(graph_csr_h_to_copy_to_d.row_ptrs, graph_csr_h.row_ptrs,
               (graph_csr_h.n + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(graph_csr_h_to_copy_to_d.col_idx, graph_csr_h.col_idx, num_edges * sizeof(int),
               hipMemcpyHostToDevice);
    hipMemcpy(graph_csr_h_to_copy_to_d.val, graph_csr_h.val, num_edges * sizeof(int),
               hipMemcpyHostToDevice);

    GraphCsr* graph_csr_d{};
    hipMalloc((void**)&graph_csr_d, sizeof(GraphCsr));
    hipMemcpy(graph_csr_d, &graph_csr_h_to_copy_to_d, sizeof(GraphCsr), hipMemcpyHostToDevice);
    int* result_d{};

    hipMalloc((void**)&result_d, adj_matrix.GetN() * sizeof(int));

    float time{};
    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    for (int i = 0; i < iters; ++i)
    {
        hipMemset(result_d, -1, adj_matrix.GetN() * sizeof(int));
        hipMemset(result_d, 0, sizeof(int));
        dim3 block_dim{SECTION_SIZE, 1, 1};
        dim3 grid_dim{
            static_cast<unsigned int>(ceil(static_cast<float>(graph_csr_h.n) / SECTION_SIZE)), 1,
            1};
        int new_vertex_visited{0};
        int curr_level{1};
        do
        {
            new_vertex_visited = 0;
            hipMemcpy(&new_vertex_visited, &new_vertex_visited, sizeof(int),
                       hipMemcpyHostToDevice);
            VertexCentricPushBFS<<<grid_dim, block_dim>>>(graph_csr_d, result_d,
                                                          &new_vertex_visited, curr_level);
            hipMemcpy(&new_vertex_visited, &new_vertex_visited, sizeof(int),
                       hipMemcpyDeviceToHost);
            ++curr_level;
        } while (new_vertex_visited != 0);

        hipMemcpy(result_h, result_d, adj_matrix.GetN() * sizeof(int), hipMemcpyDeviceToHost);
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(graph_csr_d);
    hipFree(graph_csr_h_to_copy_to_d.row_ptrs);
    hipFree(graph_csr_h_to_copy_to_d.col_idx);
    hipFree(graph_csr_h_to_copy_to_d.val);
    hipFree(result_d);

    delete[] graph_csr_h.row_ptrs;
    delete[] graph_csr_h.col_idx;
    delete[] graph_csr_h.val;

    return time;
}

float VertexCentricPullDriver(AdjacencyMatrix& adj_matrix, int* result_h, const int iters)
{
    GraphCsc graph_csc_h{adj_matrix.ToCsc()};
    GraphCsc graph_csc_h_to_copy_to_d{};
    graph_csc_h_to_copy_to_d.n = graph_csc_h.n;
    const int num_edges{graph_csc_h.col_ptrs[graph_csc_h.n]};
    hipMalloc((void**)&graph_csc_h_to_copy_to_d.col_ptrs, (graph_csc_h.n + 1) * sizeof(int));
    hipMalloc((void**)&graph_csc_h_to_copy_to_d.row_idx, num_edges * sizeof(int));
    hipMalloc((void**)&graph_csc_h_to_copy_to_d.val, num_edges * sizeof(int));
    hipMemcpy(graph_csc_h_to_copy_to_d.col_ptrs, graph_csc_h.col_ptrs,
               (graph_csc_h.n + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(graph_csc_h_to_copy_to_d.row_idx, graph_csc_h.row_idx, num_edges * sizeof(int),
               hipMemcpyHostToDevice);
    hipMemcpy(graph_csc_h_to_copy_to_d.val, graph_csc_h.val, num_edges * sizeof(int),
               hipMemcpyHostToDevice);

    GraphCsc* graph_csc_d{};
    hipMalloc((void**)&graph_csc_d, sizeof(GraphCsc));
    hipMemcpy(graph_csc_d, &graph_csc_h_to_copy_to_d, sizeof(GraphCsc), hipMemcpyHostToDevice);
    int* result_d{};

    hipMalloc((void**)&result_d, adj_matrix.GetN() * sizeof(int));

    float time{};
    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    for (int i = 0; i < iters; ++i)
    {
        hipMemset(result_d, -1, adj_matrix.GetN() * sizeof(int));
        hipMemset(result_d, 0, sizeof(int));
        dim3 block_dim{SECTION_SIZE, 1, 1};
        dim3 grid_dim{
            static_cast<unsigned int>(ceil(static_cast<float>(graph_csc_h.n) / SECTION_SIZE)), 1,
            1};
        int new_vertex_visited{0};
        int curr_level{1};
        do
        {
            new_vertex_visited = 0;
            hipMemcpy(&new_vertex_visited, &new_vertex_visited, sizeof(int),
                       hipMemcpyHostToDevice);
            VertexCentricPullBFS<<<grid_dim, block_dim>>>(graph_csc_d, result_d,
                                                          &new_vertex_visited, curr_level);
            hipMemcpy(&new_vertex_visited, &new_vertex_visited, sizeof(int),
                       hipMemcpyDeviceToHost);
            ++curr_level;
        } while (new_vertex_visited != 0);

        hipMemcpy(result_h, result_d, adj_matrix.GetN() * sizeof(int), hipMemcpyDeviceToHost);
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(graph_csc_d);
    hipFree(graph_csc_h_to_copy_to_d.col_ptrs);
    hipFree(graph_csc_h_to_copy_to_d.row_idx);
    hipFree(graph_csc_h_to_copy_to_d.val);
    hipFree(result_d);

    delete[] graph_csc_h.col_ptrs;
    delete[] graph_csc_h.row_idx;
    delete[] graph_csc_h.val;

    return time;
}

float VertexCentricPushPullDriver(AdjacencyMatrix& adj_matrix, int* result_h, const int iters)
{
    GraphCsc graph_csc_h{adj_matrix.ToCsc()};
    GraphCsc graph_csc_h_to_copy_to_d{};
    graph_csc_h_to_copy_to_d.n = graph_csc_h.n;
    const int num_edges{graph_csc_h.col_ptrs[graph_csc_h.n]};
    hipMalloc((void**)&graph_csc_h_to_copy_to_d.col_ptrs, (graph_csc_h.n + 1) * sizeof(int));
    hipMalloc((void**)&graph_csc_h_to_copy_to_d.row_idx, num_edges * sizeof(int));
    hipMalloc((void**)&graph_csc_h_to_copy_to_d.val, num_edges * sizeof(int));
    hipMemcpy(graph_csc_h_to_copy_to_d.col_ptrs, graph_csc_h.col_ptrs,
               (graph_csc_h.n + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(graph_csc_h_to_copy_to_d.row_idx, graph_csc_h.row_idx, num_edges * sizeof(int),
               hipMemcpyHostToDevice);
    hipMemcpy(graph_csc_h_to_copy_to_d.val, graph_csc_h.val, num_edges * sizeof(int),
               hipMemcpyHostToDevice);

    GraphCsc* graph_csc_d{};
    hipMalloc((void**)&graph_csc_d, sizeof(GraphCsc));
    hipMemcpy(graph_csc_d, &graph_csc_h_to_copy_to_d, sizeof(GraphCsc), hipMemcpyHostToDevice);

    GraphCsr graph_csr_h{adj_matrix.ToCsr()};
    GraphCsr graph_csr_h_to_copy_to_d{};
    graph_csr_h_to_copy_to_d.n = graph_csr_h.n;
    hipMalloc((void**)&graph_csr_h_to_copy_to_d.row_ptrs, (graph_csr_h.n + 1) * sizeof(int));
    hipMalloc((void**)&graph_csr_h_to_copy_to_d.col_idx, num_edges * sizeof(int));
    hipMalloc((void**)&graph_csr_h_to_copy_to_d.val, num_edges * sizeof(int));
    hipMemcpy(graph_csr_h_to_copy_to_d.row_ptrs, graph_csr_h.row_ptrs,
               (graph_csr_h.n + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(graph_csr_h_to_copy_to_d.col_idx, graph_csr_h.col_idx, num_edges * sizeof(int),
               hipMemcpyHostToDevice);
    hipMemcpy(graph_csr_h_to_copy_to_d.val, graph_csr_h.val, num_edges * sizeof(int),
               hipMemcpyHostToDevice);

    GraphCsr* graph_csr_d{};
    hipMalloc((void**)&graph_csr_d, sizeof(GraphCsr));
    hipMemcpy(graph_csr_d, &graph_csr_h_to_copy_to_d, sizeof(GraphCsr), hipMemcpyHostToDevice);
    int* result_d{};

    hipMalloc((void**)&result_d, adj_matrix.GetN() * sizeof(int));

    float time{};
    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    for (int i = 0; i < iters; ++i)
    {
        hipMemset(result_d, -1, adj_matrix.GetN() * sizeof(int));
        hipMemset(result_d, 0, sizeof(int));

        dim3 block_dim{SECTION_SIZE, 1, 1};
        dim3 grid_dim{
            static_cast<unsigned int>(ceil(static_cast<float>(graph_csc_h.n) / SECTION_SIZE)), 1,
            1};
        int new_vertex_visited{0};
        int total_vertices_visited{0};
        int curr_level{1};
        do
        {
            new_vertex_visited = 0;
            hipMemcpy(&new_vertex_visited, &new_vertex_visited, sizeof(int),
                       hipMemcpyHostToDevice);
            if (total_vertices_visited < adj_matrix.GetN() / 2)
            {
                VertexCentricPushBFS<<<grid_dim, block_dim>>>(graph_csr_d, result_d,
                                                              &new_vertex_visited, curr_level);
            }
            else
            {
                VertexCentricPullBFS<<<grid_dim, block_dim>>>(graph_csc_d, result_d,
                                                              &new_vertex_visited, curr_level);
            }
            hipMemcpy(&new_vertex_visited, &new_vertex_visited, sizeof(int),
                       hipMemcpyDeviceToHost);
            ++curr_level;
            total_vertices_visited += new_vertex_visited;
        } while (new_vertex_visited != 0);

        hipMemcpy(result_h, result_d, adj_matrix.GetN() * sizeof(int), hipMemcpyDeviceToHost);
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(graph_csc_d);
    hipFree(graph_csc_h_to_copy_to_d.col_ptrs);
    hipFree(graph_csc_h_to_copy_to_d.row_idx);
    hipFree(graph_csc_h_to_copy_to_d.val);
    hipFree(graph_csr_d);
    hipFree(graph_csr_h_to_copy_to_d.row_ptrs);
    hipFree(graph_csr_h_to_copy_to_d.col_idx);
    hipFree(graph_csr_h_to_copy_to_d.val);
    hipFree(result_d);

    delete[] graph_csr_h.row_ptrs;
    delete[] graph_csr_h.col_idx;
    delete[] graph_csr_h.val;
    delete[] graph_csc_h.col_ptrs;
    delete[] graph_csc_h.row_idx;
    delete[] graph_csc_h.val;

    return time;
}

float VertexCentricPushWithFrontiersDriver(AdjacencyMatrix& adj_matrix, int* result_h)
{
    GraphCsr graph_csr_h{adj_matrix.ToCsr()};
    GraphCsr graph_csr_h_to_copy_to_d{};
    graph_csr_h_to_copy_to_d.n = graph_csr_h.n;
    const int num_edges{graph_csr_h.row_ptrs[graph_csr_h.n]};
    hipMalloc((void**)&graph_csr_h_to_copy_to_d.row_ptrs, (graph_csr_h.n + 1) * sizeof(int));
    hipMalloc((void**)&graph_csr_h_to_copy_to_d.col_idx, num_edges * sizeof(int));
    hipMalloc((void**)&graph_csr_h_to_copy_to_d.val, num_edges * sizeof(int));
    hipMemcpy(graph_csr_h_to_copy_to_d.row_ptrs, graph_csr_h.row_ptrs,
               (graph_csr_h.n + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(graph_csr_h_to_copy_to_d.col_idx, graph_csr_h.col_idx, num_edges * sizeof(int),
               hipMemcpyHostToDevice);
    hipMemcpy(graph_csr_h_to_copy_to_d.val, graph_csr_h.val, num_edges * sizeof(int),
               hipMemcpyHostToDevice);

    GraphCsr* graph_csr_d{};
    hipMalloc((void**)&graph_csr_d, sizeof(GraphCsr));
    hipMemcpy(graph_csr_d, &graph_csr_h_to_copy_to_d, sizeof(GraphCsr), hipMemcpyHostToDevice);

    int* prev_frontier_d{};
    int* curr_frontier_d{};

    hipMalloc((void**)&prev_frontier_d, adj_matrix.GetN() * sizeof(int));
    hipMalloc((void**)&curr_frontier_d, adj_matrix.GetN() * sizeof(int));

    int* result_d{};

    hipMalloc((void**)&result_d, adj_matrix.GetN() * sizeof(int));

    float time{};
    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    int n_prev_frontier{1};
    int* n_curr_frontier{};
    hipMalloc((void**)&n_curr_frontier, sizeof(int));
    hipMemset(n_curr_frontier, 0, sizeof(int));
    hipMemset(prev_frontier_d, 0, sizeof(int));
    hipMemset(result_d, -1, adj_matrix.GetN() * sizeof(int));
    hipMemset(result_d, 0, sizeof(int));
    dim3 block_dim{SECTION_SIZE, 1, 1};
    dim3 grid_dim{static_cast<unsigned int>(ceil(static_cast<float>(graph_csr_h.n) / SECTION_SIZE)),
                  1, 1};
    int curr_level{1};
    do
    {
        VertexCentricPushBFSWithFrontiers<<<grid_dim, block_dim>>>(
            graph_csr_d, result_d, prev_frontier_d, curr_frontier_d, n_prev_frontier,
            n_curr_frontier, curr_level);
        hipMemcpy(&n_prev_frontier, n_curr_frontier, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(prev_frontier_d, curr_frontier_d, n_prev_frontier * sizeof(int),
                   hipMemcpyDeviceToDevice);
        hipMemset(n_curr_frontier, 0, sizeof(int));

        ++curr_level;
    } while (n_prev_frontier != 0);

    hipMemcpy(result_h, result_d, adj_matrix.GetN() * sizeof(int), hipMemcpyDeviceToHost);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(graph_csr_d);
    hipFree(graph_csr_h_to_copy_to_d.row_ptrs);
    hipFree(graph_csr_h_to_copy_to_d.col_idx);
    hipFree(graph_csr_h_to_copy_to_d.val);
    hipFree(result_d);
    hipFree(prev_frontier_d);
    hipFree(curr_frontier_d);
    hipFree(n_curr_frontier);

    delete[] graph_csr_h.row_ptrs;
    delete[] graph_csr_h.col_idx;
    delete[] graph_csr_h.val;

    return time;
}