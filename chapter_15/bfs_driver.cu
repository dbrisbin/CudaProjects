#include "hip/hip_runtime.h"
/// @file bfs_driver.cu
/// @brief Definition of driver function declared in bfs_driver.h.

#include <stdio.h>
#include <algorithm>
#include "bfs.h"
#include "bfs_driver.h"
#include "types/adjacency_matrix.h"
#include "types/constants.h"
#include "types/graph_coo.h"

float BfsDriver(AdjacencyMatrix& adj_matrix, int* result_h, const int iters,
                const BfsKernel kernel_to_use)
{
    GraphCoo graph_coo_h{adj_matrix.ToCoo()};
    GraphCoo* graph_coo_d{};
    hipMalloc((void**)&graph_coo_d, sizeof(GraphCoo));
    hipMemcpy(graph_coo_d, &graph_coo_h, sizeof(GraphCoo), hipMemcpyHostToDevice);
    hipMalloc((void**)&graph_coo_d->src, graph_coo_h.num_edges * sizeof(int));
    hipMalloc((void**)&graph_coo_d->dst, graph_coo_h.num_edges * sizeof(int));
    hipMalloc((void**)&graph_coo_d->val, graph_coo_h.num_edges * sizeof(int));
    hipMemcpy(graph_coo_d->src, graph_coo_h.src, graph_coo_h.num_edges * sizeof(int),
               hipMemcpyHostToDevice);
    hipMemcpy(graph_coo_d->dst, graph_coo_h.dst, graph_coo_h.num_edges * sizeof(int),
               hipMemcpyHostToDevice);
    hipMemcpy(graph_coo_d->val, graph_coo_h.val, graph_coo_h.num_edges * sizeof(int),
               hipMemcpyHostToDevice);
    int* result_d{};

    hipMalloc((void**)&result_d, adj_matrix.GetNumNnz() * sizeof(int));
    hipMemset(result_d, -1, adj_matrix.GetNumNnz() * sizeof(int));
    hipMemset(result_d, 0, sizeof(int));

    switch (kernel_to_use)
    {
        case BfsKernel::kEdgeCentric:
        {
            for (int i = 0; i < iters; ++i)
            {
                dim3 block_dim{SECTION_SIZE, 1, 1};
                dim3 grid_dim{static_cast<unsigned int>(
                                  ceil(static_cast<float>(adj_matrix.GetNumNnz()) / SECTION_SIZE)),
                              1, 1};
                int new_vertex_visited{0};
                int curr_level{1};
                do
                {
                    new_vertex_visited = 0;
                    hipMemcpy(&new_vertex_visited, &new_vertex_visited, sizeof(int),
                               hipMemcpyHostToDevice);
                    EdgeCentricBFS<<<grid_dim, block_dim>>>(graph_coo_d, result_d,
                                                            &new_vertex_visited, curr_level);
                    hipMemcpy(&new_vertex_visited, &new_vertex_visited, sizeof(int),
                               hipMemcpyDeviceToHost);
                    ++curr_level;
                } while (new_vertex_visited != 0);

                hipMemcpy(result_h, result_d, adj_matrix.GetNumNnz() * sizeof(int),
                           hipMemcpyDeviceToHost);
            }
            break;
        }
        case BfsKernel::kNumKernels:
        default:
        {
            printf("Invalid kernel type\n");
            return -1.0f;
        }
    }

    hipFree(graph_coo_d->src);
    hipFree(graph_coo_d->dst);
    hipFree(graph_coo_d->val);
    hipFree(graph_coo_d);
    hipFree(result_d);

    float total_time{0.0f};
    return total_time;
}
