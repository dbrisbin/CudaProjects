#include <stdio.h>
#include "parallel_scan.h"
#include "parallel_scan_driver.h"
#include "types/constants.h"

extern "C" float ParallelScanDriver(const ParallelScanDataType* data_h,
                                    ParallelScanDataType* result_h, const unsigned int length,
                                    const enum parallelScanKernelToUse kernel_to_use,
                                    const int iters)
{
    ParallelScanDataType* data_d;
    ParallelScanDataType* result_d;
    ParallelScanDataType* end_vals_d;
    int* flags_d;

    dim3 dim_block, dim_grid;

    hipMalloc((void**)&data_d, length * sizeof(ParallelScanDataType));
    hipMalloc((void**)&result_d, length * sizeof(ParallelScanDataType));
    hipMalloc((void**)&end_vals_d, length / SECTION_SIZE / CFACTOR * sizeof(ParallelScanDataType));
    hipMalloc((void**)&flags_d, length / SECTION_SIZE / CFACTOR * sizeof(int));

    hipMemcpy(data_d, data_h, length * sizeof(ParallelScanDataType), hipMemcpyHostToDevice);

    float time;
    float total_time = 0.0f;
    hipEvent_t start, stop;

    for (int iter = 0; iter < iters; ++iter)
    {
        dim_block = dim3(SECTION_SIZE, 1, 1);
        dim_grid = dim3(length / SECTION_SIZE / CFACTOR, 1, 1);
        ResetArray<<<dim_grid, dim_block>>>(flags_d, length / SECTION_SIZE / CFACTOR, 0);
        hipMemset(&block_counter, 0, sizeof(ParallelScanDataType));
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        switch (kernel_to_use)
        {
            case kKoggeStoneInclusive:
                dim_block = dim3(SECTION_SIZE, 1, 1);
                dim_grid = dim3(1, 1, 1);
                KoggeStoneInclusiveKernel<<<dim_grid, dim_block>>>(data_d, result_d, length);
                break;
            case kKoggeStoneExclusive:
                dim_block = dim3(SECTION_SIZE, 1, 1);
                dim_grid = dim3(1, 1, 1);
                KoggeStoneExclusiveKernel<<<dim_grid, dim_block>>>(data_d, result_d, length);
                break;
            case kKoggeStoneDoubleBufferingInclusive:
                dim_block = dim3(SECTION_SIZE, 1, 1);
                dim_grid = dim3(1, 1, 1);
                KoggeStoneDoubleBufferingInclusiveKernel<<<dim_grid, dim_block>>>(data_d, result_d,
                                                                                  length);
                break;
            case kKoggeStoneDoubleBufferingExclusive:
                dim_block = dim3(SECTION_SIZE, 1, 1);
                dim_grid = dim3(1, 1, 1);
                KoggeStoneDoubleBufferingExclusiveKernel<<<dim_grid, dim_block>>>(data_d, result_d,
                                                                                  length);
                break;
            case kBrentKungInclusive:
                dim_block = dim3(SECTION_SIZE, 1, 1);
                dim_grid = dim3(1, 1, 1);
                BrentKungInclusiveKernel<<<dim_grid, dim_block>>>(data_d, result_d, length);
                break;
            case kBrentKungExclusive:
                dim_block = dim3(SECTION_SIZE, 1, 1);
                dim_grid = dim3(1, 1, 1);
                BrentKungExclusiveKernel<<<dim_grid, dim_block>>>(data_d, result_d, length);
                break;
            case kCoarseningInclusive:
                dim_block = dim3(SECTION_SIZE, 1, 1);
                dim_grid = dim3(1, 1, 1);
                ThreadCoarseningInclusiveKernel<<<dim_grid, dim_block>>>(data_d, result_d, length);
                break;
            case kCoarseningExclusive:
                dim_block = dim3(SECTION_SIZE, 1, 1);
                dim_grid = dim3(1, 1, 1);
                ThreadCoarseningExclusiveKernel<<<dim_grid, dim_block>>>(data_d, result_d, length);
                break;
            case kCoarseningSegmented:
                // Phase 1:
                dim_block = dim3(SECTION_SIZE, 1, 1);
                dim_grid = dim3(length / SECTION_SIZE / CFACTOR, 1, 1);
                ThreadCoarseningSegmentedScanKernelPhase1<<<dim_grid, dim_block>>>(
                    data_d, result_d, end_vals_d, length);

                // Phase 2:
                dim_block = dim3(SECTION_SIZE, 1, 1);
                dim_grid = dim3(1, 1, 1);
                ThreadCoarseningInclusiveKernel<<<dim_grid, dim_block>>>(
                    end_vals_d, end_vals_d, length / SECTION_SIZE / CFACTOR);

                // Phase 3:
                dim_block = dim3(SECTION_SIZE, 1, 1);
                dim_grid = dim3(length / SECTION_SIZE / CFACTOR, 1, 1);
                ThreadCoarseningSegmentedScanKernelPhase3<<<dim_grid, dim_block>>>(
                    result_d, end_vals_d, length);
                break;
            case kStreaming:
                dim_block = dim3(SECTION_SIZE, 1, 1);
                dim_grid = dim3(length / SECTION_SIZE / CFACTOR, 1, 1);
                StreamingKernel<<<dim_grid, dim_block>>>(data_d, result_d, flags_d, end_vals_d,
                                                         length);
                break;
            case kNumKernels:
            default:
                printf("Invalid kernel selected! Try again!\n");
                return -1.0;
        }

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        total_time += time;
    }

    hipError_t err = hipMemcpy(result_h, result_d, length * sizeof(ParallelScanDataType),
                                 hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d.\n", hipGetErrorString(err), __FILE__, __LINE__);
    }

    hipFree(result_d);
    hipFree(data_d);
    hipFree(end_vals_d);

    return total_time;
}
