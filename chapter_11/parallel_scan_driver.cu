#include <stdio.h>
#include "parallel_scan.h"
#include "parallel_scan_driver.h"
#include "types/constants.h"

extern "C" float ParallelScanDriver(const ParallelScanDataType* data_h,
                                    ParallelScanDataType* result_h, const unsigned int length,
                                    const enum parallelScanKernelToUse kernel_to_use,
                                    const int iters)
{
    ParallelScanDataType* data_d;
    ParallelScanDataType* result_d;
    dim3 dimBlock, dimGrid;

    hipMalloc((void**)&data_d, length * sizeof(ParallelScanDataType));
    hipMalloc((void**)&result_d, length * sizeof(ParallelScanDataType));

    hipMemcpy(data_d, data_h, length * sizeof(ParallelScanDataType), hipMemcpyHostToDevice);

    float time;
    float total_time = 0.0f;
    hipEvent_t start, stop;

    for (int iter = 0; iter < iters; ++iter)
    {

        switch (kernel_to_use)
        {
            case kKoggeStone:
                dimBlock = dim3(SECTION_SIZE, 1, 1);
                dimGrid = dim3(1, 1, 1);
                hipEventCreate(&start);
                hipEventCreate(&stop);
                hipEventRecord(start, 0);
                KoggeStoneKernel<<<dimGrid, dimBlock>>>(data_d, result_d, length);
                break;
            case kKoggeStoneDoubleBuffering:
                dimBlock = dim3(SECTION_SIZE, 1, 1);
                dimGrid = dim3(1, 1, 1);
                hipEventCreate(&start);
                hipEventCreate(&stop);
                hipEventRecord(start, 0);
                KoggeStoneDoubleBufferingKernel<<<dimGrid, dimBlock>>>(data_d, result_d, length);
                break;
            case kBrentKung:
                dimBlock = dim3(SECTION_SIZE, 1, 1);
                dimGrid = dim3(1, 1, 1);
                hipEventCreate(&start);
                hipEventCreate(&stop);
                hipEventRecord(start, 0);
                BrentKungKernel<<<dimGrid, dimBlock>>>(data_d, result_d, length);
                break;
            case kNumKernels:
            default:
                break;
        }
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        total_time += time;
    }

    hipError_t err = hipMemcpy(result_h, result_d, length * sizeof(ParallelScanDataType),
                                 hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d.\n", hipGetErrorString(err), __FILE__, __LINE__);
    }

    hipFree(result_d);
    hipFree(data_d);

    return total_time;
}
