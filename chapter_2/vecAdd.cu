#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

void vecAddOnHost(float *A_h, float *B_h, float *C_h, int n)
{
    for (int i = 0; i < n; ++i)
    {
        C_h[i] = A_h[i] + B_h[i];
    }
}

__global__ void vecAddKernel(float *A, float *B, float *C, int n)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n)
    {
        C[i] = A[i] + B[i];
    }
}

void vecAddOnDevice(float *A_h, float *B_h, float *C_h, int n)
{
    int size = n * sizeof(float);
    float *A_d, *B_d, *C_d;

    hipMalloc((void **)&A_d, size);
    hipMalloc((void **)&B_d, size);
    hipMalloc((void **)&C_d, size);

    hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

    vecAddKernel<<<ceil(n / 256.0), 256.0>>>(A_d, B_d, C_d, n);

    hipError_t err = hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d.", hipGetErrorString(err), __FILE__, __LINE__);
    }
    hipFree(C_d);
    hipFree(B_d);
    hipFree(A_d);
}

int main()
{
    float *A, *B, *C, *D;
    int n;

    printf("Enter number of elements:");
    scanf("%d", &n);

    A = (float *)malloc(n * sizeof(float));
    B = (float *)malloc(n * sizeof(float));
    C = (float *)malloc(n * sizeof(float));
    D = (float *)malloc(n * sizeof(float));

    for (int i = 0; i < n; ++i)
    {
        A[i] = i;
        B[i] = (n - i) * 2;
    }

    vecAddOnHost(A, B, C, n);
    vecAddOnDevice(A, B, D, n);

    printf("A: [");
    for (int i = 0; i < n; ++i)
    {
        printf("%0.1f, ", A[i]);
    }
    printf("]\n");

    printf("B: [");
    for (int i = 0; i < n; ++i)
    {
        printf("%0.1f, ", B[i]);
    }
    printf("]\n");

    printf("C: [");
    for (int i = 0; i < n; ++i)
    {
        printf("%0.1f, ", C[i]);
    }
    printf("]\n");

    printf("D: [");
    for (int i = 0; i < n; ++i)
    {
        printf("%0.1f, ", D[i]);
    }
    printf("]\n");
}
