#include "hip/hip_runtime.h"
#include <iostream>
#include "fhd.h"
#include "fhd_driver.h"

float FhdDriver(const float* r_phi, const float* r_d, const float* i_phi, const float* i_d,
                const float* x, const float* k_x, const float* y, const float* k_y, const float* z,
                const float* k_z, const int M, const int N, float* r_fhd, float* i_fhd,
                const FhdKernels kernel_to_use, const int iters)
{
    // copy data to device,
    float *d_r_phi, *d_r_d, *d_i_phi, *d_i_d, *d_x, *d_k_x, *d_y, *d_k_y, *d_z, *d_k_z, *d_r_mu,
        *d_i_mu, *d_r_fhd, *d_i_fhd;
    hipMalloc((void**)&d_r_phi, M * sizeof(float));
    hipMalloc((void**)&d_r_d, M * sizeof(float));
    hipMalloc((void**)&d_i_phi, M * sizeof(float));
    hipMalloc((void**)&d_i_d, M * sizeof(float));
    hipMalloc((void**)&d_k_x, M * sizeof(float));
    hipMalloc((void**)&d_k_y, M * sizeof(float));
    hipMalloc((void**)&d_k_z, M * sizeof(float));
    hipMalloc((void**)&d_x, N * sizeof(float));
    hipMalloc((void**)&d_y, N * sizeof(float));
    hipMalloc((void**)&d_z, N * sizeof(float));

    hipMalloc((void**)&d_r_mu, M * sizeof(float));
    hipMalloc((void**)&d_i_mu, M * sizeof(float));
    hipMalloc((void**)&d_r_fhd, N * sizeof(float));
    hipMalloc((void**)&d_i_fhd, N * sizeof(float));

    hipMemcpy(d_r_phi, r_phi, M * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_r_d, r_d, M * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_i_phi, i_phi, M * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_i_d, i_d, M * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_k_x, k_x, M * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_k_y, k_y, M * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_k_z, k_z, M * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_z, z, N * sizeof(float), hipMemcpyHostToDevice);

    // Start timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (int i = 0; i < iters; i++)
    {
        hipMemset(d_r_fhd, 0, N * sizeof(float));
        hipMemset(d_i_fhd, 0, N * sizeof(float));

        switch (kernel_to_use)
        {
            case FhdKernels::kBasic:
            {
                dim3 block_dim(SECTION_SIZE, 1, 1);
                dim3 grid_dim(std::ceil(static_cast<float>(M) / SECTION_SIZE), 1, 1);

                BasicKernel<<<grid_dim, block_dim>>>(d_r_phi, d_r_d, d_i_phi, d_i_d, d_x, d_k_x,
                                                     d_y, d_k_y, d_z, d_k_z, d_r_mu, d_i_mu, M, N,
                                                     d_r_fhd, d_i_fhd);
                break;
            }
            case FhdKernels::kLoopInterchangeBasic:
            {
                dim3 block_dim(SECTION_SIZE, 1, 1);
                dim3 grid_dim(std::ceil(static_cast<float>(M) / SECTION_SIZE), 1, 1);
                ComputeMu<<<grid_dim, block_dim>>>(d_r_phi, d_r_d, d_i_phi, d_i_d, d_r_mu, d_i_mu,
                                                   M);

                grid_dim.x = std::ceil(static_cast<float>(N) / SECTION_SIZE);
                ComputeFHDWithNThreads<<<grid_dim, block_dim>>>(
                    d_x, d_k_x, d_y, d_k_y, d_z, d_k_z, d_r_mu, d_i_mu, M, N, d_r_fhd, d_i_fhd);
                break;
            }
            case FhdKernels::kLoopInterchangeWithRegisters:
            {
                dim3 block_dim(SECTION_SIZE, 1, 1);
                dim3 grid_dim(std::ceil(static_cast<float>(M) / SECTION_SIZE), 1, 1);
                ComputeMu<<<grid_dim, block_dim>>>(d_r_phi, d_r_d, d_i_phi, d_i_d, d_r_mu, d_i_mu,
                                                   M);

                grid_dim.x = std::ceil(static_cast<float>(N) / SECTION_SIZE);
                ComputeFHDWithNThreadsAndRegisters<<<grid_dim, block_dim>>>(
                    d_x, d_k_x, d_y, d_k_y, d_z, d_k_z, d_r_mu, d_i_mu, M, N, d_r_fhd, d_i_fhd);
                break;
            }
            case FhdKernels::kLoopInterchangeWithRegistersAndRestrict:
            {
                dim3 block_dim(SECTION_SIZE, 1, 1);
                dim3 grid_dim(std::ceil(static_cast<float>(M) / SECTION_SIZE), 1, 1);
                ComputeMu<<<grid_dim, block_dim>>>(d_r_phi, d_r_d, d_i_phi, d_i_d, d_r_mu, d_i_mu,
                                                   M);

                grid_dim.x = std::ceil(static_cast<float>(N) / SECTION_SIZE);
                ComputeFHDWithNThreadsRegistersAndRestrict<<<grid_dim, block_dim>>>(
                    d_x, d_k_x, d_y, d_k_y, d_z, d_k_z, d_r_mu, d_i_mu, M, N, d_r_fhd, d_i_fhd);
                break;
            }
            case FhdKernels::kLoopInterchangeWithRegistersAndConstantMem:
            {
                dim3 block_dim(SECTION_SIZE, 1, 1);
                dim3 grid_dim(std::ceil(static_cast<float>(M) / SECTION_SIZE), 1, 1);
                ComputeMu<<<grid_dim, block_dim>>>(d_r_phi, d_r_d, d_i_phi, d_i_d, d_r_mu, d_i_mu,
                                                   M);

                grid_dim.x = std::ceil(static_cast<float>(N) / SECTION_SIZE);
                for (size_t i{0}; i < M / kChunkSize; i++)
                {
                    const auto M_offset = i * kChunkSize;
                    if (hipMemcpyToSymbol(HIP_SYMBOL(k_x_c), &k_x[M_offset], kChunkSize * sizeof(float)) ||
                        hipMemcpyToSymbol(HIP_SYMBOL(k_y_c), &k_y[M_offset], kChunkSize * sizeof(float)) ||
                        hipMemcpyToSymbol(HIP_SYMBOL(k_z_c), &k_z[M_offset], kChunkSize * sizeof(float)))
                    {
                        std::cerr << "Error copying to constant memory" << std::endl;
                        return -1.F;
                    }

                    ComputeFHDWithNThreadsRegistersAndConstantMem<<<grid_dim, block_dim>>>(
                        d_x, d_y, d_z, d_r_mu, d_i_mu, static_cast<int>(kChunkSize), N,
                        static_cast<int>(M_offset), d_r_fhd, d_i_fhd);
                }
                if (M % kChunkSize != 0)
                {
                    const auto M_offset = kChunkSize * (M / kChunkSize);
                    const int remaining = M % kChunkSize;
                    if (hipMemcpyToSymbol(HIP_SYMBOL(k_x_c), &k_x[M_offset], remaining * sizeof(float)) ||
                        hipMemcpyToSymbol(HIP_SYMBOL(k_y_c), &k_y[M_offset], remaining * sizeof(float)) ||
                        hipMemcpyToSymbol(HIP_SYMBOL(k_z_c), &k_z[M_offset], remaining * sizeof(float)))
                    {
                        std::cerr << "Error copying to constant memory" << std::endl;
                        return -1.F;
                    }

                    ComputeFHDWithNThreadsRegistersAndConstantMem<<<grid_dim, block_dim>>>(
                        d_x, d_y, d_z, d_r_mu, d_i_mu, remaining, N, static_cast<int>(M_offset),
                        d_r_fhd, d_i_fhd);
                }
                break;
            }
            case FhdKernels::kNumKernels:
            default:
                std::cerr << "Invalid kernel to use" << std::endl;
                break;
        }
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(r_fhd, d_r_fhd, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(i_fhd, d_i_fhd, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_r_phi);
    hipFree(d_r_d);
    hipFree(d_i_phi);
    hipFree(d_i_d);
    hipFree(d_x);
    hipFree(d_k_x);
    hipFree(d_y);
    hipFree(d_k_y);
    hipFree(d_z);
    hipFree(d_k_z);
    hipFree(d_r_fhd);
    hipFree(d_i_fhd);

    return milliseconds;
}