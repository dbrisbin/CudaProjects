#include <stdio.h>
#include "reduction.h"
#include "reductionDriver.h"
#include "types/constants.h"

extern "C" float reductionDriver(const ReductionDataType* data_h, const int length,
                                 ReductionDataType* result_h,
                                 const enum reductionKernelToUse kernel_to_use, const int iters)
{
    ReductionDataType* data_d;
    ReductionDataType* result_d;
    dim3 dimBlock, dimGrid;

    hipMalloc((void**)&data_d, length * sizeof(ReductionDataType));
    hipMalloc((void**)&result_d, sizeof(ReductionDataType));

    float time;
    float total_time = 0.0f;
    hipEvent_t start, stop;

    for (int iter = 0; iter < iters; ++iter)
    {
        hipMemcpy(data_d, data_h, length * sizeof(ReductionDataType), hipMemcpyHostToDevice);

        switch (kernel_to_use)
        {
            case kBasic:
                dimBlock = dim3(TILE_WIDTH, 1, 1);
                dimGrid = dim3(1, 1, 1);
                hipEventCreate(&start);
                hipEventCreate(&stop);
                hipEventRecord(start, 0);
                basicReduction<<<dimGrid, dimBlock>>>(data_d, length, result_d);
                break;
            case kCoalescing:
            case kSharedMemory:
            case kSegmented:
            case kCoarsening:
            case kNumKernels:
            default:
                break;
        }
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        total_time += time;
    }

    hipError_t err =
        hipMemcpy(result_h, result_d, sizeof(ReductionDataType), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d.\n", hipGetErrorString(err), __FILE__, __LINE__);
    }

    hipFree(result_d);
    hipFree(data_d);

    return total_time;
}
