#include <stdio.h>
#include "reduction.h"
#include "reductionDriver.h"
#include "types/constants.h"

extern "C" float reductionDriver(const ReductionDataType* data_h, const int length,
                                 ReductionDataType* result_h,
                                 const enum reductionKernelToUse kernel_to_use, const int iters)
{
    ReductionDataType* data_d;
    ReductionDataType* result_d;
    ReductionDataType identity = reductionIdentity();
    dim3 dimBlock, dimGrid;

    hipMalloc((void**)&data_d, length * sizeof(ReductionDataType));
    hipMalloc((void**)&result_d, sizeof(ReductionDataType));

    float time;
    float total_time = 0.0f;
    hipEvent_t start, stop;

    for (int iter = 0; iter < iters; ++iter)
    {
        hipMemcpy(data_d, data_h, length * sizeof(ReductionDataType), hipMemcpyHostToDevice);
        hipMemcpy(result_d, &identity, sizeof(ReductionDataType), hipMemcpyHostToDevice);

        switch (kernel_to_use)
        {
            case kBasic:
                dimBlock = dim3(TILE_WIDTH, 1, 1);
                dimGrid = dim3(1, 1, 1);
                hipEventCreate(&start);
                hipEventCreate(&stop);
                hipEventRecord(start, 0);
                basicReduction<<<dimGrid, dimBlock>>>(data_d, length, result_d);
                break;
            case kCoalescing:
                dimBlock = dim3(TILE_WIDTH, 1, 1);
                dimGrid = dim3(1, 1, 1);
                hipEventCreate(&start);
                hipEventCreate(&stop);
                hipEventRecord(start, 0);
                coalescingReduction<<<dimGrid, dimBlock>>>(data_d, length, result_d);
                break;
            case kCoalescingModified:
                dimBlock = dim3(TILE_WIDTH, 1, 1);
                dimGrid = dim3(1, 1, 1);
                hipEventCreate(&start);
                hipEventCreate(&stop);
                hipEventRecord(start, 0);
                coalescingReduction<<<dimGrid, dimBlock>>>(data_d, length, result_d);
                break;

            case kSharedMemory:
                dimBlock = dim3(TILE_WIDTH, 1, 1);
                dimGrid = dim3(1, 1, 1);
                hipEventCreate(&start);
                hipEventCreate(&stop);
                hipEventRecord(start, 0);
                sharedMemoryReduction<<<dimGrid, dimBlock>>>(data_d, length, result_d);
                break;
            case kSegmented:
                dimBlock = dim3(TILE_WIDTH, 1, 1);
                dimGrid = dim3(1, 1, 1);
                hipEventCreate(&start);
                hipEventCreate(&stop);
                hipEventRecord(start, 0);
                segmentedReduction<<<dimGrid, dimBlock>>>(data_d, length, result_d);
                break;
            case kCoarsening:
                dimBlock = dim3(TILE_WIDTH, 1, 1);
                dimGrid = dim3(1, 1, 1);
                hipEventCreate(&start);
                hipEventCreate(&stop);
                hipEventRecord(start, 0);
                coarseningReduction<<<dimGrid, dimBlock>>>(data_d, length, result_d);
                break;
            case kNumKernels:
            default:
                break;
        }
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        total_time += time;
    }

    hipError_t err =
        hipMemcpy(result_h, result_d, sizeof(ReductionDataType), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d.\n", hipGetErrorString(err), __FILE__, __LINE__);
    }

    hipFree(result_d);
    hipFree(data_d);

    return total_time;
}
