#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "dcs.h"
#include "types/constants.h"
#include "types/types.h"
#include "utils.h"

__constant__ Atom atoms_c[kChunkSize];

__global__ void DcsScatter(float* energy_grid, const dim3 grid_size, const float spacing,
                           const float z, const unsigned int num_atoms)
{
    const unsigned int n{blockIdx.x * blockDim.x + threadIdx.x};
    if (n >= num_atoms)
    {
        return;
    }
    const Atom& atom = atoms_c[n];
    const float dz = z - atom.z;
    const float dz2 = dz * dz;

    for (unsigned int j{0U}; j < grid_size.y; ++j)
    {
        const float dy = j * spacing - atom.y;
        const float dy2 = dy * dy;
        for (unsigned int i{0U}; i < grid_size.x; ++i)
        {
            const float dx = i * spacing - atom.x;
            const float dx2 = dx * dx;
            const float r2 = dx2 + dy2 + dz2;
            atomicAdd(&energy_grid[LinearizeIndex(i, j, static_cast<int>(grid_size.x))],
                      atom.charge / sqrtf(r2));
        }
    }
}

__global__ void DcsGatherBasic(float* energy_grid, const dim3 grid_size, const float spacing,
                               const float z, const unsigned int num_atoms)
{
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i >= grid_size.x || j >= grid_size.y)
    {
        return;
    }
    float energy{0.0f};
    const float x = i * spacing;
    const float y = j * spacing;

    for (unsigned int k{0U}; k < num_atoms; ++k)
    {
        const Atom& atom = atoms_c[k];
        const float dx = x - atom.x;
        const float dy = y - atom.y;
        const float dz = z - atom.z;
        const float r2 = dx * dx + dy * dy + dz * dz;
        energy += atom.charge / sqrtf(r2);
    }
    energy_grid[LinearizeIndex(i, j, static_cast<int>(grid_size.x))] += energy;
}

__global__ void DcsGatherCoarsened(float* energy_grid, const dim3 grid_size, const float spacing,
                                   const float z, const unsigned int num_atoms)
{
    const unsigned int i = (blockIdx.x * blockDim.x + threadIdx.x) * kCoarseningFactor;
    const unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i >= grid_size.x || j >= grid_size.y)
    {
        return;
    }
    float energy[kCoarseningFactor];
    for (unsigned int k{0U}; k < kCoarseningFactor; ++k)
    {
        energy[k] = 0.0f;
    }

    const float x = i * spacing;
    const float y = j * spacing;

    for (unsigned int k{0U}; k < num_atoms; ++k)
    {
        const Atom& atom = atoms_c[k];

        float dx = x - atom.x;
        const float dy = y - atom.y;
        const float dz = z - atom.z;
        const float dy2pdz2 = dy * dy + dz * dz;

        for (unsigned int l{0U}; l < kCoarseningFactor; ++l)
        {
            energy[l] += atom.charge / sqrtf(dx * dx + dy2pdz2);
            dx += spacing;
        }
    }
    const int idx = LinearizeIndex(i, j, static_cast<int>(grid_size.x));
    for (unsigned int k{0U}; k < kCoarseningFactor; ++k)
    {
        if (i + k < grid_size.x)
        {
            energy_grid[idx + k] += energy[k];
        }
    }
}

__global__ void DcsGatherCoarsenedCoalesced(float* energy_grid, const dim3 grid_size,
                                            const float spacing, const float z,
                                            const unsigned int num_atoms)
{
    const unsigned int i = blockIdx.x * blockDim.x * kCoarseningFactor + threadIdx.x;
    const unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i >= grid_size.x || j >= grid_size.y)
    {
        return;
    }
    float energy[kCoarseningFactor];
    for (unsigned int k{0U}; k < kCoarseningFactor; ++k)
    {
        energy[k] = 0.0f;
    }

    const float x = i * spacing;
    const float y = j * spacing;

    for (unsigned int k{0U}; k < num_atoms; ++k)
    {
        const Atom& atom = atoms_c[k];

        float dx = x - atom.x;
        const float dy = y - atom.y;
        const float dz = z - atom.z;
        const float dy2pdz2 = dy * dy + dz * dz;

        for (unsigned int l{0U}; l < kCoarseningFactor; ++l)
        {
            energy[l] += atom.charge / sqrtf(dx * dx + dy2pdz2);
            dx += blockDim.x * spacing;
        }
    }
    const int idx = LinearizeIndex(i, j, static_cast<int>(grid_size.x));
    for (unsigned int k{0U}; k < kCoarseningFactor; ++k)
    {
        if (i + blockDim.x * k < grid_size.x)
        {
            energy_grid[idx + blockDim.x * k] += energy[k];
        }
    }
}
